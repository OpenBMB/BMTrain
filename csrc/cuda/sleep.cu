#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cstdint>
#include <assert.h>
#include "watch_dog.hpp"

extern WatchDog* watchDog;

static __global__ void spin_kernel(clock_t count) {

    clock_t start = clock();
    clock_t now = start;
    #pragma unroll(1) // avoid instruct optimization.
    while ((now - start < count) || (now + (0xffffffff - start) < count)) {
        now = clock();
    }
}

void checkCUDAStatus(hipError_t err) {
    if (err == hipSuccess)
        return;
    throw std::runtime_error(
        std::string("CUDA Error: ") + hipGetErrorString(err));
}

void cuda_spin(int ms, std::uintptr_t stream) {
    int dev, clockRate = 0;
    checkCUDAStatus(hipGetDevice(&dev));
    checkCUDAStatus(hipDeviceGetAttribute(&clockRate, hipDeviceAttributeClockRate, dev));
    CUDAEvent event(reinterpret_cast<hipStream_t>(stream), "cuda_sleep");
    event.recordStart();
    spin_kernel<<<1, 1, 0, reinterpret_cast<hipStream_t>(stream)>>>(ms * clock_t(clockRate));
    checkCUDAStatus(hipGetLastError());
    event.recordEnd();
    watchDog->watch(event);
}