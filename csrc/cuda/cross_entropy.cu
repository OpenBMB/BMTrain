#include "hip/hip_runtime.h"
#include <hip/hip_fp16.h>
#include <torch/extension.h>
#include <ATen/cuda/HIPContext.h>
#include "reduce.cuh"

namespace {
// blocks <m>,      threads<1024>
__global__ void cross_entropy_forward(
    int64_t n,
    const half *input,      // (m, n)
    const int32_t *target,  // (m)
    half *softmax,          // (m, n)
    float *output,          // (m)
    int32_t ignore_index
) {
    int64_t base_idx = blockIdx.x * n;

    float local_max = -INFINITY;
    for (int64_t i = threadIdx.x; i < n; i += blockDim.x) {
        local_max = fmaxf(__half2float(input[base_idx + i]), local_max);
    }

    local_max = fmaxf(block_allreduce_max(local_max), -1e6);
    
    float local_sum = 0;
    for (int64_t i = threadIdx.x; i < n; i += blockDim.x) {
        local_sum += expf(__half2float(input[base_idx + i]) - local_max);
    }
    local_sum = block_allreduce_sum(local_sum) + 1e-10; // avoid nan
    
    for (int64_t i = threadIdx.x; i < n; i += blockDim.x) {
        softmax[base_idx + i] = __float2half( expf(__half2float(input[base_idx + i]) - local_max) / local_sum );
    }

    if (threadIdx.x == 0) {
        if (target[blockIdx.x] != ignore_index) {
            output[blockIdx.x] = -__half2float(input[base_idx + target[blockIdx.x]]) + local_max + logf(local_sum);
        } else {
            output[blockIdx.x] = 0;
        }
    }
}

// blocks <m>,      threads<1024>
__global__ void cross_entropy_backward(
    int64_t n,
    const float *grad_output,   // (m)
    const int32_t *target,      // (m)
    const half *softmax,        // (m, n)
    half *grad_input,           // (m, n)
    int32_t ignore_index
) {
    int64_t base_idx = blockIdx.x * n;

    int32_t t = target[blockIdx.x];
    if (t == ignore_index) {
        half v = __float2half(0.);
        for (int64_t i = threadIdx.x; i < n; i += blockDim.x) {
            grad_input[base_idx + i] = v;
        }
    }
    else {
        half v = __float2half(grad_output[blockIdx.x]);
        for (int64_t i = threadIdx.x; i < n; i += blockDim.x) {
            grad_input[base_idx + i] = i==t ? __hsub(__hmul(softmax[base_idx + i], v), v) : __hmul(softmax[base_idx + i], v);
        }
    }
}

// blocks <m>,      threads<1024>
__global__ void cross_entropy_forward_inplace(
    int64_t n,
    half *x,                // (m, n)
    const int32_t *target,  // (m)
    float *output,          // (m)
    int32_t ignore_index
) {
    int64_t base_idx = blockIdx.x * n;

    float local_max = -INFINITY;
    for (int64_t i = threadIdx.x; i < n; i += blockDim.x) {
        local_max = fmaxf(__half2float(x[base_idx + i]), local_max);
    }
    local_max = fmaxf(block_allreduce_max(local_max), -1e6);
    
    float local_sum = 0;
    for (int64_t i = threadIdx.x; i < n; i += blockDim.x) {
        local_sum += expf(__half2float(x[base_idx + i]) - local_max);
    }
    local_sum = block_allreduce_sum(local_sum) + 1e-10; // avoid nan

    if (threadIdx.x == 0) {
        if (target[blockIdx.x] != ignore_index) {
            output[blockIdx.x] = -__half2float(x[base_idx + target[blockIdx.x]]) + local_max + logf(local_sum);
        } else {
            output[blockIdx.x] = 0;
        }
    }

    __syncthreads();
    
    for (int64_t i = threadIdx.x; i < n; i += blockDim.x) {
        x[base_idx + i] = __float2half( expf(__half2float(x[base_idx + i]) - local_max) / local_sum );
    }
}

// blocks <m>,      threads<1024>
__global__ void cross_entropy_backward_inplace(
    int64_t n,
    const float *grad_output,   // (m)
    const int32_t *target,      // (m)
    half *x,                    // (m, n)
    int32_t ignore_index
) {
    int64_t base_idx = blockIdx.x * n;

    int32_t t = target[blockIdx.x];
    if (t == ignore_index) {
        half v = __float2half(0.);
        for (int64_t i = threadIdx.x; i < n; i += blockDim.x) {
            x[base_idx + i] = v;
        }
    }
    else {
        half v = __float2half(grad_output[blockIdx.x]);
        __syncthreads();
        for (int64_t i = threadIdx.x; i < n; i += blockDim.x) {
            x[base_idx + i] = i==t ? __hsub(__hmul(x[base_idx + i], v), v) : __hmul(x[base_idx + i], v);
        }
    }
}

}

void cross_entropy_forward_launcher(
    int32_t m, int32_t n,
    const torch::Tensor &input,
    const torch::Tensor &target,
    torch::Tensor &softmax,
    torch::Tensor &output,
    int32_t ignore_index
) {
    auto input_ptr = reinterpret_cast<half*>(input.data_ptr<at::Half>());
    auto target_ptr = target.data_ptr<int32_t>();
    auto softmax_ptr = reinterpret_cast<half*>(softmax.data_ptr<at::Half>());
    auto output_ptr = output.data_ptr<float>();
    int32_t threads = 1024;
    auto stream = at::cuda::getCurrentCUDAStream();
    cross_entropy_forward<<<m, threads, 0, stream.stream()>>>(n, input_ptr, target_ptr, softmax_ptr, output_ptr, ignore_index);
}

void cross_entropy_backward_launcher(
    int32_t m, int32_t n,
    const torch::Tensor &grad_output,
    const torch::Tensor &target,
    const torch::Tensor &softmax,
    torch::Tensor &grad_input,
    int32_t ignore_index
) {
    auto output_ptr = grad_output.data_ptr<float>();
    auto target_ptr = target.data_ptr<int32_t>();
    auto softmax_ptr = reinterpret_cast<half*>(softmax.data_ptr<at::Half>());
    auto input_ptr = reinterpret_cast<half*>(grad_input.data_ptr<at::Half>());
    int32_t threads = 1024;
    auto stream = at::cuda::getCurrentCUDAStream();
    cross_entropy_backward<<<m, threads, 0, stream.stream()>>>(n, output_ptr, target_ptr, softmax_ptr, input_ptr, ignore_index);
}

void cross_entropy_forward_inplace_launcher(
    int32_t m, int32_t n,
    torch::Tensor &x,
    const torch::Tensor &target,
    torch::Tensor &output,
    int32_t ignore_index
) {
    auto x_ptr = reinterpret_cast<half*>(x.data_ptr<at::Half>());
    auto target_ptr = target.data_ptr<int32_t>();
    auto output_ptr = output.data_ptr<float>();
    int32_t threads = 1024;
    auto stream = at::cuda::getCurrentCUDAStream();
    cross_entropy_forward_inplace<<<m, threads, 0, stream.stream()>>>(n, x_ptr, target_ptr, output_ptr, ignore_index);
}

void cross_entropy_backward_inplace_launcher(
    int32_t m, int32_t n,
    const torch::Tensor &grad_output,
    const torch::Tensor &target,
    torch::Tensor &x,
    int32_t ignore_index
) {
    auto output_ptr = grad_output.data_ptr<float>();
    auto target_ptr = target.data_ptr<int32_t>();
    auto x_ptr = reinterpret_cast<half*>(x.data_ptr<at::Half>());
    int32_t threads = 1024;
    auto stream = at::cuda::getCurrentCUDAStream();
    cross_entropy_backward_inplace<<<m, threads, 0, stream.stream()>>>(n, output_ptr, target_ptr, x_ptr, ignore_index);
}