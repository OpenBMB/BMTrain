#include "hip/hip_runtime.h"
#include "reduce.cuh"
#include <cstdint>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include "bfloat16.cuh"

namespace {
// blocks <m>,      threads<1024>
__global__ void cross_entropy_forward_fp16(
    int64_t n,
    const half *input,      // (m, n)
    const int32_t *target,  // (m)
    half *softmax,          // (m, n)
    float *output,          // (m)
    int32_t ignore_index
) {
    int64_t base_idx = blockIdx.x * n;

    float local_max = -INFINITY;
    for (int64_t i = threadIdx.x; i < n; i += blockDim.x) {
        local_max = fmaxf(__half2float(input[base_idx + i]), local_max);
    }

    local_max = fmaxf(block_allreduce_max(local_max), -1e6);
    
    float local_sum = 0;
    for (int64_t i = threadIdx.x; i < n; i += blockDim.x) {
        local_sum += expf(__half2float(input[base_idx + i]) - local_max);
    }
    local_sum = block_allreduce_sum(local_sum) + 1e-10; // avoid nan
    
    for (int64_t i = threadIdx.x; i < n; i += blockDim.x) {
        softmax[base_idx + i] = __float2half( expf(__half2float(input[base_idx + i]) - local_max) / local_sum );
    }

    if (threadIdx.x == 0) {
        if (target[blockIdx.x] != ignore_index) {
            output[blockIdx.x] = -__half2float(input[base_idx + target[blockIdx.x]]) + local_max + logf(local_sum);
        } else {
            output[blockIdx.x] = 0;
        }
    }
}

// blocks <m>,      threads<1024>
__global__ void cross_entropy_backward_inplace_fp16(
    int64_t n,
    const float *grad_output,   // (m)
    const int32_t *target,      // (m)
    half *x,                    // (m, n)
    int32_t ignore_index
) {
    int64_t base_idx = blockIdx.x * n;

    int32_t t = target[blockIdx.x];
    float v = grad_output[blockIdx.x];
    for (int64_t i = threadIdx.x; i < n; i += blockDim.x) {
        x[base_idx + i] = __float2half(i==t ? (__half2float(x[base_idx + i])-1)*v : __half2float(x[base_idx + i])*v);
    }
}

// blocks <m>,      threads<1024>
__global__ void cross_entropy_forward_bf16(
    int64_t n,
    const std::uintptr_t input_ptr,      // (m, n)
    const int32_t *target,  // (m)
    std::uintptr_t softmax_ptr,          // (m, n)
    float *output,          // (m)
    int32_t ignore_index
) {
#ifdef BF16_SUPPORT
    const __hip_bfloat16* input = reinterpret_cast<const __hip_bfloat16*>(input_ptr);
    __hip_bfloat16* softmax = reinterpret_cast<__hip_bfloat16*>(softmax_ptr);
    int64_t base_idx = blockIdx.x * n;

    float local_max = -INFINITY;
    for (int64_t i = threadIdx.x; i < n; i += blockDim.x) {
        local_max = fmaxf(__bfloat162float(input[base_idx + i]), local_max);
    }

    local_max = fmaxf(block_allreduce_max(local_max), -1e6);
    
    float local_sum = 0;
    for (int64_t i = threadIdx.x; i < n; i += blockDim.x) {
        local_sum += expf(__bfloat162float(input[base_idx + i]) - local_max);
    }
    local_sum = block_allreduce_sum(local_sum) + 1e-10; // avoid nan
    
    for (int64_t i = threadIdx.x; i < n; i += blockDim.x) {
        softmax[base_idx + i] = __float2bfloat16( expf(__bfloat162float(input[base_idx + i]) - local_max) / local_sum );
    }

    if (threadIdx.x == 0) {
        if (target[blockIdx.x] != ignore_index) {
            output[blockIdx.x] = -__bfloat162float(input[base_idx + target[blockIdx.x]]) + local_max + logf(local_sum);
        } else {
            output[blockIdx.x] = 0;
        }
    }
#endif
}

// blocks <m>,      threads<1024>
__global__ void cross_entropy_backward_inplace_bf16(
    int64_t n,
    const float *grad_output,   // (m)
    const int32_t *target,      // (m)
    std::uintptr_t x_ptr,                    // (m, n)
    int32_t ignore_index
) {
#ifdef BF16_SUPPORT
    __hip_bfloat16* x = reinterpret_cast<__hip_bfloat16*>(x_ptr);
    int64_t base_idx = blockIdx.x * n;

    int32_t t = target[blockIdx.x];
    float v = grad_output[blockIdx.x];
    for (int64_t i = threadIdx.x; i < n; i += blockDim.x) {
        x[base_idx + i] = __float2bfloat16(i==t ? (__bfloat162float(x[base_idx + i])-1)*v : __bfloat162float(x[base_idx + i])*v);
    }
#endif
}

// blocks <m>,      threads<1024>
__global__ void fused_sumexp_fp16(
    int64_t n,
    const half *input,              // (m, n)
    const float *global_max,        // (m)
    float *global_sum               // (m)
) {
    int64_t base_idx = blockIdx.x * n;
    float local_max = global_max[blockIdx.x];
    
    float local_sum = 0;
    for (int64_t i = threadIdx.x; i < n; i += blockDim.x) {
        local_sum += expf(__half2float(input[base_idx + i]) - local_max);
    }
    local_sum = block_allreduce_sum(local_sum);
    if (threadIdx.x == 0) {
        global_sum[blockIdx.x] = local_sum;
    }
}

// blocks <m>,      threads<1024>
__global__ void fused_sumexp_bf16(
    int64_t n,
    const std::uintptr_t input_ptr,              // (m, n)
    const float *global_max,        // (m)
    float *global_sum               // (m)
) {
#ifdef BF16_SUPPORT
    const __hip_bfloat16* input = reinterpret_cast<const __hip_bfloat16*>(input_ptr);
    int64_t base_idx = blockIdx.x * n;
    float local_max = global_max[blockIdx.x];
    
    float local_sum = 0;
    for (int64_t i = threadIdx.x; i < n; i += blockDim.x) {
        local_sum += expf(__bfloat162float(input[base_idx + i]) - local_max);
    }
    local_sum = block_allreduce_sum(local_sum);
    if (threadIdx.x == 0) {
        global_sum[blockIdx.x] = local_sum;
    }
#endif
}

// blocks <m>,      threads<1024>
__global__ void fused_softmax_inplace_fp16(
    int64_t n,
    half *softmax,                  // (m, n)
    const float *global_max,        // (m)
    const float *global_sum         // (m)
) {
    int64_t base_idx = blockIdx.x * n;
    float local_max = global_max[blockIdx.x];
    float local_sum = global_sum[blockIdx.x];
    
    for (int64_t i = threadIdx.x; i < n; i += blockDim.x) {
        softmax[base_idx + i] = __float2half( expf(__half2float(softmax[base_idx + i]) - local_max) / local_sum );
    }
}

// blocks <m>,      threads<1024>
__global__ void fused_softmax_inplace_bf16(
    int64_t n,
    std::uintptr_t softmax_ptr,                  // (m, n)
    const float *global_max,        // (m)
    const float *global_sum         // (m)
) {
#ifdef BF16_SUPPORT
    __hip_bfloat16* softmax = reinterpret_cast<__hip_bfloat16*>(softmax_ptr);
    int64_t base_idx = blockIdx.x * n;
    float local_max = global_max[blockIdx.x];
    float local_sum = global_sum[blockIdx.x];
    
    for (int64_t i = threadIdx.x; i < n; i += blockDim.x) {
        softmax[base_idx + i] = __float2bfloat16( expf(__bfloat162float(softmax[base_idx + i]) - local_max) / local_sum );
    }
#endif
}
}

void cross_entropy_forward_fp16_launcher(
    int32_t m, int32_t n,
    std::uintptr_t input,
    std::uintptr_t target,
    std::uintptr_t softmax,
    std::uintptr_t output,
    int32_t ignore_index,
    std::uintptr_t stream
) {
    auto input_ptr = reinterpret_cast<half*>(input);
    auto target_ptr = reinterpret_cast<int32_t*>(target);
    auto softmax_ptr = reinterpret_cast<half*>(softmax);
    auto output_ptr = reinterpret_cast<float*>(output);
    int32_t threads = 1024;
    cross_entropy_forward_fp16<<<m, threads, 0, reinterpret_cast<hipStream_t>(stream)>>>(n, input_ptr, target_ptr, softmax_ptr, output_ptr, ignore_index);
}

void cross_entropy_backward_inplace_fp16_launcher(
    int32_t m, int32_t n,
    std::uintptr_t grad_output,
    std::uintptr_t target,
    std::uintptr_t x,
    int32_t ignore_index,
    std::uintptr_t stream
) {
    auto output_ptr = reinterpret_cast<float*>(grad_output);
    auto target_ptr = reinterpret_cast<int32_t*>(target);
    auto x_ptr = reinterpret_cast<half*>(x);
    int32_t threads = 1024;
    cross_entropy_backward_inplace_fp16<<<m, threads, 0, reinterpret_cast<hipStream_t>(stream)>>>(n, output_ptr, target_ptr, x_ptr, ignore_index);
}

void cross_entropy_forward_bf16_launcher(
    int32_t m, int32_t n,
    std::uintptr_t input,
    std::uintptr_t target,
    std::uintptr_t softmax,
    std::uintptr_t output,
    int32_t ignore_index,
    std::uintptr_t stream
) {
    auto target_ptr = reinterpret_cast<int32_t*>(target);
    auto output_ptr = reinterpret_cast<float*>(output);
    int32_t threads = 1024;
    cross_entropy_forward_bf16<<<m, threads, 0, reinterpret_cast<hipStream_t>(stream)>>>(n, input, target_ptr, softmax, output_ptr, ignore_index);
}

void cross_entropy_backward_inplace_bf16_launcher(
    int32_t m, int32_t n,
    std::uintptr_t grad_output,
    std::uintptr_t target,
    std::uintptr_t x,
    int32_t ignore_index,
    std::uintptr_t stream
) {
    auto output_ptr = reinterpret_cast<float*>(grad_output);
    auto target_ptr = reinterpret_cast<int32_t*>(target);
    int32_t threads = 1024;
    cross_entropy_backward_inplace_bf16<<<m, threads, 0, reinterpret_cast<hipStream_t>(stream)>>>(n, output_ptr, target_ptr, x, ignore_index);
}

void fused_sumexp_fp16_launcher(
    int32_t m, int32_t n,
    std::uintptr_t logits,
    std::uintptr_t max_logits,
    std::uintptr_t sum_exp_logits,
    std::uintptr_t stream
) {
    auto logits_ptr = reinterpret_cast<half*>(logits);
    auto max_logits_ptr = reinterpret_cast<float*>(max_logits);
    auto sum_exp_logits_ptr = reinterpret_cast<float*>(sum_exp_logits);
    int32_t threads = 1024;
    fused_sumexp_fp16<<<m, threads, 0, reinterpret_cast<hipStream_t>(stream)>>>(n, logits_ptr, max_logits_ptr, sum_exp_logits_ptr);
}

void fused_sumexp_bf16_launcher(
    int32_t m, int32_t n,
    std::uintptr_t logits,
    std::uintptr_t max_logits,
    std::uintptr_t sum_exp_logits,
    std::uintptr_t stream
) {
    auto max_logits_ptr = reinterpret_cast<float*>(max_logits);
    auto sum_exp_logits_ptr = reinterpret_cast<float*>(sum_exp_logits);
    int32_t threads = 1024;
    fused_sumexp_bf16<<<m, threads, 0, reinterpret_cast<hipStream_t>(stream)>>>(n, logits, max_logits_ptr, sum_exp_logits_ptr);
}

void fused_softmax_inplace_fp16_launcher(
    int32_t m, int32_t n,
    std::uintptr_t logits,
    std::uintptr_t max_logits,
    std::uintptr_t sum_exp_logits,
    std::uintptr_t stream
) {
    auto logits_ptr = reinterpret_cast<half*>(logits);
    auto max_logits_ptr = reinterpret_cast<float*>(max_logits);
    auto sum_exp_logits_ptr = reinterpret_cast<float*>(sum_exp_logits);
    int32_t threads = 1024;
    fused_softmax_inplace_fp16<<<m, threads, 0, reinterpret_cast<hipStream_t>(stream)>>>(n, logits_ptr, max_logits_ptr, sum_exp_logits_ptr);
}

void fused_softmax_inplace_bf16_launcher(
    int32_t m, int32_t n,
    std::uintptr_t logits,
    std::uintptr_t max_logits,
    std::uintptr_t sum_exp_logits,
    std::uintptr_t stream
) {
    auto max_logits_ptr = reinterpret_cast<float*>(max_logits);
    auto sum_exp_logits_ptr = reinterpret_cast<float*>(sum_exp_logits);
    int32_t threads = 1024;
    fused_softmax_inplace_bf16<<<m, threads, 0, reinterpret_cast<hipStream_t>(stream)>>>(n, logits, max_logits_ptr, sum_exp_logits_ptr);
}